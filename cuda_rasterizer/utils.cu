#include "hip/hip_runtime.h"
#include "utils.h"
#include "auxiliary.h"


// Equation (9) in "3D Gaussian Splatting as Markov Chain Monte Carlo"
__global__ void compute_relocation(
    int P, 
    float* opacity_old, 
    float* scale_old, 
    int* N, 
    float* binoms, 
    int n_max, 
    float* opacity_new, 
    float* scale_new) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= P) return;
    
    int N_idx = N[idx];
    float denom_sum = 0.0f;

    // compute new opacity
    opacity_new[idx] = 1.0f - powf(1.0f - opacity_old[idx], 1.0f / N_idx);
    
    // compute new scale
    for (int i = 1; i <= N_idx; ++i) {
        for (int k = 0; k <= (i-1); ++k) {
            float bin_coeff = binoms[(i-1) * n_max + k];
            float term = (pow(-1, k) / sqrt(k + 1)) * pow(opacity_new[idx], k + 1);
            denom_sum += (bin_coeff * term);
        }
    }
    float coeff = (opacity_old[idx] / denom_sum);
    for (int i = 0; i < 3; ++i)
        scale_new[idx * 3 + i] = coeff * scale_old[idx * 3 + i];
}

void UTILS::ComputeRelocation(
    int P,
    float* opacity_old,
    float* scale_old,
    int* N,
    float* binoms,
    int n_max,
    float* opacity_new,
    float* scale_new)
{
	int num_blocks = (P + 255) / 256;
	dim3 block(256, 1, 1);
	dim3 grid(num_blocks, 1, 1);
	compute_relocation<<<grid, block>>>(P, opacity_old, scale_old, N, binoms, n_max, opacity_new, scale_new);
}